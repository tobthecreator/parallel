#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <string.h>

// Error checking macro
#define cudaCheckError()                                             \
    {                                                                \
        hipError_t e = hipGetLastError();                          \
        if (e != hipSuccess)                                        \
        {                                                            \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, \
                   hipGetErrorString(e));                           \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

// Kernel declaration
__global__ void vectorAdd(float *a, float *b, float *c, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

int main()
{
    // Print device properties
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, 0);
    if (err != hipSuccess)
    {
        printf("Failed to get device properties: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }
    printf("Device Name: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);

    // Test vector addition
    const int N = 1000;
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_a = (float *)malloc(size);
    float *h_b = (float *)malloc(size);
    float *h_c = (float *)malloc(size);

    if (h_a == NULL || h_b == NULL || h_c == NULL)
    {
        printf("Failed to allocate host memory\n");
        return EXIT_FAILURE;
    }

    // Initialize host arrays
    for (int i = 0; i < N; i++)
    {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    err = hipMalloc(&d_a, size);
    if (err != hipSuccess)
    {
        printf("Failed to allocate device memory for d_a: %s\n", hipGetErrorString(err));
        free(h_a);
        free(h_b);
        free(h_c);
        return EXIT_FAILURE;
    }

    err = hipMalloc(&d_b, size);
    if (err != hipSuccess)
    {
        printf("Failed to allocate device memory for d_b: %s\n", hipGetErrorString(err));
        hipFree(d_a);
        free(h_a);
        free(h_b);
        free(h_c);
        return EXIT_FAILURE;
    }

    err = hipMalloc(&d_c, size);
    if (err != hipSuccess)
    {
        printf("Failed to allocate device memory for d_c: %s\n", hipGetErrorString(err));
        hipFree(d_a);
        hipFree(d_b);
        free(h_a);
        free(h_b);
        free(h_c);
        return EXIT_FAILURE;
    }

    // Copy inputs to device
    err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("Failed to copy data to device: %s\n", hipGetErrorString(err));
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        free(h_a);
        free(h_b);
        free(h_c);
        return EXIT_FAILURE;
    }

    err = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("Failed to copy data to device: %s\n", hipGetErrorString(err));
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        free(h_a);
        free(h_b);
        free(h_c);
        return EXIT_FAILURE;
    }

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    cudaCheckError(); // Check for kernel launch errors

    // Synchronize to wait for kernel completion
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        printf("Kernel execution failed: %s\n", hipGetErrorString(err));
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        free(h_a);
        free(h_b);
        free(h_c);
        return EXIT_FAILURE;
    }

    // Copy result back to host
    err = hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy result from device: %s\n", hipGetErrorString(err));
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        free(h_a);
        free(h_b);
        free(h_c);
        return EXIT_FAILURE;
    }

    // Verify
    bool success = true;
    for (int i = 0; i < N; i++)
    {
        if (fabs(h_c[i] - 3.0f) > 1e-5f)
        {
            printf("Verification failed at index %d: %f != 3.0\n", i, h_c[i]);
            success = false;
            break;
        }
    }
    if (success)
        printf("CUDA test passed successfully!\n");

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return success ? EXIT_SUCCESS : EXIT_FAILURE;
}